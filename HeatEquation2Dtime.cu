#include <stdio.h>
#include <malloc.h>
#include <hip/hip_runtime.h>
#define Ntime 50 
#define M 64
#define N 64
#define dx 0.1
#define dy 0.1
#define dt 0.1
#define D 0.01
#define BlockSizeX 16 
#define BlockSizeY 16 
#define GridSizeX 4 // 64:16 = 128
#define GridSizeY 4 // 64:16 = 128

void printGrid(float *T)
{
    int i,j;
    for (i=0;i<M;i++)
    {
        for (j=0;j<N;j++)
        {
            printf("%.2f ", *(T+i*N+j));
        }
        printf("\n");
    }
}

void initialize(float *T)
{
    int i,j;
    for (i=0;i<M;i++)
    {
        for (j=0;j<N;j++)
        {
            if ((i>=(M/2-4*(M/64)))&&(i<(M/2+4*(M/64)))&&(j>=(N/2-4*(N/64)))&&(j<(N/2+4*(N/64))))
                *(T+i*N+j) = 100.0;
            else
                *(T+i*N+j) = 25.0;
        }
    }
}

//Biến đổi phương trình đạo hàm riêng thành phương trình vi phân toàn phần (ODE) 

__global__ void Derivative(float *T, float *dT)
{
    float c,left,right,up,down;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= 1 && i < M - 1 && j >= 1 && j < N - 1) {
        up = *(T+(i-1)*N+j);
        down = *(T+(i+1)*N+j);
        left = *(T+i*N+j-1);
        right = *(T+i*N+j+1);
        c = *(T+i*N+j);
        *(dT+i*N+j) = D*(((up-2*c+down)/(dx*dx))+((left-2*c+right)/(dy*dy)));
    }
}
 
//Giải phương trình đã biến đổi (bằng euler thuận)

__global__ void SolvingODE(float *T, float *dT)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < M && j < N) {
            *(T+i*N+j) = *(T+i*N+j) + dt*(*(dT+i*N+j));
    }
}

int main() 
{
    float *Tcpu;
    Tcpu = (float*)malloc((M*N)*sizeof(float));
    initialize(Tcpu);
    // CUDA code
    // 1.Declare, allocate mem
    float *Tgpu, *dTgpu; 
    hipMalloc((void**)&Tgpu, (M*N)*sizeof(float));
    hipMalloc((void**)&dTgpu, (M*N)*sizeof(float));
    // 2.Copy input from CPU to GPU
    hipMemcpy(Tgpu, Tcpu, (M*N)*sizeof(float), hipMemcpyHostToDevice);
    // 3.Define Block and Threads Structures 
    dim3 dimGrid(GridSizeX, GridSizeY);
    dim3 dimBlock(BlockSizeX, BlockSizeY);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    for (int t=0; t<Ntime; t++) {
        Derivative<<<dimGrid,dimBlock>>>(Tgpu,dTgpu);
        SolvingODE<<<dimGrid,dimBlock>>>(Tgpu,dTgpu);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("GPU Time: %f ms\n", milliseconds);
    hipMemcpy(Tcpu, Tgpu, M*N * sizeof(float), hipMemcpyDeviceToHost);
    // Giải phóng bộ nhớ
    free(Tcpu);
    hipFree(Tgpu);
    hipFree(dTgpu);
    return 0;
}
